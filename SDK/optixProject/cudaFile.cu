#include "hip/hip_runtime.h"

#include "optixProject.h"
#include <optixu/optixu_aabb.h>

#define WC_PREFIX __device__
#define WC_NO_FILES
#include "woven_cloth.cpp"
#include "random.h"

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

rtDeclareVariable(float3, dPdu, attribute dPdu, );
rtDeclareVariable(float3, dPdv, attribute dPdv, );

//Shadow ray variables
rtDeclareVariable(unsigned int, shadow_samples, , );
rtDeclareVariable(unsigned int, light_radius, , );

//Anti-aliasing variables
rtDeclareVariable(float,		jitter_factor, , );
rtDeclareVariable(unsigned int, frame, , );
rtBuffer<unsigned int, 2>		rnd_seeds;

//
// Pinhole camera implementation
//
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;

RT_PROGRAM void pinhole_camera()
{
    size_t2 screen = output_buffer.size();
	float3 prd_result;
	PerRayData_radiance prd;


	unsigned int seed = rot_seed(rnd_seeds[launch_index], frame);

	for (int i = 0; i < 1; i++) {
		float2 subpixel_jitter = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f) * jitter_factor;

		float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
		float3 ray_origin = eye;
		float3 ray_direction = normalize(d.x*U + d.y*V + W);

		optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);


		prd.ray_id = launch_index;
		prd.importance = 1.f;
		prd.depth = 0;

		rtTrace(top_object, ray, prd);
		
		prd_result += prd.result;
		if (i > 0) {
			prd_result = prd_result / 2;
		}
		
		
	}
	output_buffer[launch_index] = make_color(prd_result);
}


//
// Environment map background
//
rtDeclareVariable(float3,bg_color, , );
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
    if(prd_radiance.depth == 0){
        prd_radiance.result = bg_color;
    }else{
        float theta = atan2f( ray.direction.x, ray.direction.z );
        float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
        float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
        float v     = 0.5f * ( 1.0f + sin(phi) );
        prd_radiance.result = make_float3( tex2D(envmap, u, v) );
    }
}


//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_shadow()
{
    // this material is opaque, so it fully attenuates all shadow rays
    prd_shadow.attenuation = make_float3(0);

    rtTerminateRay();
}


//
// Procedural metal surface shader
//

/*
* Translated to CUDA C from Larry Gritz's LGRustyMetal.sl shader found at:
* http://renderman.org/RMR/Shaders/LGShaders/LGRustyMetal.sl
*
* Used with permission from tal AT renderman DOT org.
*/

rtDeclareVariable(float3,   ambient_light_color, , );
rtBuffer<BasicLight>        lights;   
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float,    importance_cutoff, , );      
rtDeclareVariable(int,      max_depth, , );
rtDeclareVariable(float3,   reflectivity_n, , );

rtDeclareVariable(float, metalKa, , ) = 1;
rtDeclareVariable(float, metalKs, , ) = 1;
rtDeclareVariable(float, metalroughness, , ) = .1;
rtDeclareVariable(float3, metalcolor, , ) = {.7, .7, .7};
rtDeclareVariable(float, txtscale, , ) = .02;
#define MAXOCTAVES 6

rtTextureSampler<float, 3> noise_texture;
static __device__ __inline__ float snoise(float3 p)
{
    return tex3D(noise_texture, p.x, p.y, p.z) * 2 -1;
}

RT_PROGRAM void metal_closest_hit_radiance()
{
    float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
    float3 hit_point = ray.origin + t_hit * ray.direction;

    /* Sum several octaves of abs(snoise), i.e. turbulence.  Limit the
    * number of octaves by the estimated change in PP between adjacent
    * shading samples.
    */
    float3 PP = txtscale * hit_point;
    float a = 1;
    float sum = 0;
    for(int i = 0; i < MAXOCTAVES; i++ ){
        sum += a * fabs(snoise(PP));
        PP *= 2.0f;
        a *= 0.5f;
    }


    float3 color = metalcolor * metalKa * ambient_light_color;
    for(int i = 0; i < lights.size(); ++i) {
        BasicLight light = lights[i];
        float3 L = normalize(light.pos - hit_point);
        float nmDl = dot( ffnormal, L);

        if( nmDl > 0.0f){
            // cast shadow ray
            PerRayData_shadow shadow_prd;
            shadow_prd.attenuation = make_float3(1.0f);
            float Ldist = length(light.pos - hit_point);
            optix::Ray shadow_ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
            rtTrace(top_shadower, shadow_ray, shadow_prd);
            float3 light_attenuation = shadow_prd.attenuation;

            if( fmaxf(light_attenuation) > 0.0f ){
                float3 Lc = light.color * light_attenuation;
                color += Lc;

                float r = nmDl;
                if(nmDl > 0.0f){
                    float3 H = normalize(L - ray.direction);
                    float nmDh = dot( ffnormal, H );
                    if(nmDh > 0)
                        color += r * metalKs * Lc * pow(nmDh, 1.f/metalroughness);
                }
            }

        }
    }

    float3 r = schlick(-dot(ffnormal, ray.direction), reflectivity_n );
    float importance = prd_radiance.importance * optix::luminance( r );

    // reflection ray
    if( importance > importance_cutoff && prd_radiance.depth < max_depth) {
        PerRayData_radiance refl_prd;
        refl_prd.importance = importance;
        refl_prd.depth = prd_radiance.depth+1;
        float3 R = reflect( ray.direction, ffnormal );
        optix::Ray refl_ray( hit_point, R, radiance_ray_type, scene_epsilon );
        rtTrace(top_object, refl_ray, refl_prd);
        color += r * refl_prd.result;
    }

    prd_radiance.result = color;
}


//
// Phong surface shading with shadows and schlick-approximated fresnel reflections.
// Uses procedural texture to determine diffuse response.
//
rtDeclareVariable(float,  phong_exp, , );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kd, , );

RT_PROGRAM void chair_closest_hit_radiance()
{
	
    float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
    float3 color = Ka * ambient_light_color;
	
	
	float shadow_intensity = 0.3f/(float)shadow_samples;
	unsigned int num_lights = lights.size();
	PerRayData_shadow shadow_prd;
    
    float3 hit_point = ray.origin + t_hit * ray.direction;

    
    

    for(int i = 0; i < lights.size(); ++i) {
        BasicLight light = lights[i];
        float3 L = normalize(light.pos - hit_point);
        float nDl = dot( ffnormal, L);

		for(int i=0;i<shadow_samples;i++){
			float r = light_radius*sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +0,
				(prd_radiance.ray_id.y*shadow_samples +i)*3 +0, 8);
			float theta = sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +1,
				(prd_radiance.ray_id.y*shadow_samples +i)*3 +1, 8);
			float phi   = sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +2, 
				(prd_radiance.ray_id.y*shadow_samples +i)*3 +2, 8);
			float3 offset = make_float3(r*sin(theta)*sin(phi),r*cos(theta)*sin(phi),r*cos(phi));
			float Ldist = optix::length(light.pos + offset - hit_point);
			float3 L = optix::normalize(light.pos + offset - hit_point);
        
			// cast shadow ray
			if ( light.casts_shadow ) {
				shadow_prd.attenuation = make_float3(1.0f);
				optix::Ray shadow_ray = optix::make_Ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
				rtTrace(top_shadower, shadow_ray, shadow_prd);
				color = shadow_prd.attenuation*shadow_intensity*color + (1.f-shadow_intensity)*color;
			}
		}
    }

    float3 r = schlick(-dot(ffnormal, ray.direction), reflectivity_n);
    float importance = prd_radiance.importance * optix::luminance( r );

    // reflection ray
    if( importance > importance_cutoff && prd_radiance.depth < max_depth) {
        PerRayData_radiance refl_prd;
        refl_prd.importance = importance;
        refl_prd.depth = prd_radiance.depth+1;
        float3 R = reflect( ray.direction, ffnormal );
        optix::Ray refl_ray( hit_point, R, radiance_ray_type, scene_epsilon );
        rtTrace(top_object, refl_ray, refl_prd);
        color += r * refl_prd.result;
    }


    prd_radiance.result = color;
}


//
// Bounding box program for programmable convex hull primitive
//
rtDeclareVariable(float3, chull_bbmin, , );
rtDeclareVariable(float3, chull_bbmax, , );
RT_PROGRAM void chull_bounds (int primIdx, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = chull_bbmin;
    aabb->m_max = chull_bbmax;
}


//
// Intersection program for programmable convex hull primitive
//
rtBuffer<float4> planes;
RT_PROGRAM void chull_intersect(int primIdx)
{
    int n = planes.size();
    float t0 = -FLT_MAX;
    float t1 = FLT_MAX;
    float3 t0_normal = make_float3(0);
    float3 t1_normal = make_float3(0);
    for(int i = 0; i < n && t0 < t1; ++i ) {
        float4 plane = planes[i];
        float3 n = make_float3(plane);
        float  d = plane.w;

        float denom = dot(n, ray.direction);
        float t = -(d + dot(n, ray.origin))/denom;
        if( denom < 0){
            // enter
            if(t > t0){
                t0 = t;
                t0_normal = n;
            }
        } else {
            //exit
            if(t < t1){
                t1 = t;
                t1_normal = n;
            }
        }
    }

    if(t0 > t1)
        return;

    if(rtPotentialIntersection( t0 )){
        shading_normal = geometric_normal = t0_normal;
        rtReportIntersection(0);
    } else if(rtPotentialIntersection( t1 )){
        shading_normal = geometric_normal = t1_normal;
        rtReportIntersection(0);
    }
}


//
// (NEW)
// Attenuates shadow rays for shadowing transparent objects
//

rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void glass_any_hit_shadow()
{
    float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float nDi = fabs(dot(world_normal, ray.direction));

    prd_shadow.attenuation *= 1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1));

    rtIgnoreIntersection();
}


//
// Dielectric surface shader
//
rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(int,          refraction_maxdepth, , );
rtDeclareVariable(int,          reflection_maxdepth, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );
RT_PROGRAM void glass_closest_hit_radiance()
{
    // intersection vectors
    const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
    const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
    const float3 i = ray.direction;                                            // incident direction

    float reflection = 1.0f;
    float3 result = make_float3(0.0f);

    float3 beer_attenuation;
    if(dot(n, ray.direction) > 0){
        // Beer's law attenuation
        beer_attenuation = exp(extinction_constant * t_hit);
    } else {
        beer_attenuation = make_float3(1);
    }

    // refraction
    if (prd_radiance.depth < min(refraction_maxdepth, max_depth))
    {
        float3 t;                                                            // transmission direction
        if ( refract(t, i, n, refraction_index) )
        {

            // check for external or internal reflection
            float cos_theta = dot(i, n);
            if (cos_theta < 0.0f)
                cos_theta = -cos_theta;
            else
                cos_theta = dot(t, n);

            reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

            float importance = prd_radiance.importance * (1.0f-reflection) * optix::luminance( refraction_color * beer_attenuation );
            if ( importance > importance_cutoff ) {
                optix::Ray ray( h, t, radiance_ray_type, scene_epsilon );
                PerRayData_radiance refr_prd;
                refr_prd.depth = prd_radiance.depth+1;
                refr_prd.importance = importance;

                rtTrace( top_object, ray, refr_prd );
                result += (1.0f - reflection) * refraction_color * refr_prd.result;
            } else {
                result += (1.0f - reflection) * refraction_color * cutoff_color;
            }
        }
        // else TIR
    }

    // reflection
    if (prd_radiance.depth < min(reflection_maxdepth, max_depth))
    {
        float3 r = reflect(i, n);

        float importance = prd_radiance.importance * reflection * optix::luminance( reflection_color * beer_attenuation );
        if ( importance > importance_cutoff ) {
            optix::Ray ray( h, r, radiance_ray_type, scene_epsilon );
            PerRayData_radiance refl_prd;
            refl_prd.depth = prd_radiance.depth+1;
            refl_prd.importance = importance;

            rtTrace( top_object, ray, refl_prd );
            result += reflection * reflection_color * refl_prd.result;
        } else {
            result += reflection * reflection_color * cutoff_color;
        }
    }

    result = result * beer_attenuation;

    prd_radiance.result = result;
}


//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
    //output_buffer[launch_index] = make_color( bad_color );
}

// ------------- Only Shadows ---------------

RT_PROGRAM void only_shadows_closest_hit_radiance()
{
    float shadow_intensity = 0.3f/(float)shadow_samples;
    float3 color = bg_color;
    float3 hit_point = ray.origin + t_hit * ray.direction;
    unsigned int num_lights = lights.size();
    for(int i = 0; i < num_lights; ++i) {
        BasicLight light = lights[i];
        for(int i=0;i<shadow_samples;i++){
            float r = light_radius*sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +0,
                (prd_radiance.ray_id.y*shadow_samples +i)*3 +0, 8);
            float theta = sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +1,
                (prd_radiance.ray_id.y*shadow_samples +i)*3 +1, 8);
            float phi   = sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +2, 
                (prd_radiance.ray_id.y*shadow_samples +i)*3 +2, 8);
            float3 offset = make_float3(r*sin(theta)*sin(phi),r*cos(theta)*sin(phi),r*cos(phi));
            float Ldist = optix::length(light.pos + offset - hit_point);
            float3 L = optix::normalize(light.pos + offset - hit_point);
        
            // cast shadow ray
            if ( light.casts_shadow ) {
                PerRayData_shadow shadow_prd;
                shadow_prd.attenuation = make_float3(1.0f);
                optix::Ray shadow_ray = optix::make_Ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
                rtTrace(top_shadower, shadow_ray, shadow_prd);
                color = shadow_prd.attenuation*shadow_intensity*color + (1.f-shadow_intensity)*color;
            }
        }
    }

    prd_radiance.result = color;
}

// ------------- Woven Cloth ----------------

rtDeclareVariable(wcWeaveParameters, wc_parameters, , );
rtDeclareVariable(float, wc_specular_strength, , );
rtDeclareVariable(PatternEntry, wc_pattern, , );

RT_PROGRAM void cloth_closest_hit_radiance()
{
    float3 hit_point = ray.origin + t_hit * ray.direction;
    wcWeaveParameters params = wc_parameters;
    params.pattern_entry = &wc_pattern;

    float3 color = make_float3(0.f);
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 p_normal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
    float3 u_vec = normalize( rtTransformVector( RT_OBJECT_TO_WORLD, dPdu));
    float3 v_vec = normalize( rtTransformVector( RT_OBJECT_TO_WORLD, dPdv));

    wcIntersectionData intersection;
    intersection.uv_x = texcoord.x;
    intersection.uv_y = texcoord.y;
    intersection.wo_x = optix::dot(-ray.direction, u_vec);
    intersection.wo_y = optix::dot(-ray.direction, v_vec);
    intersection.wo_z = optix::dot(-ray.direction, p_normal);

	
    PerRayData_shadow shadow_prd;
	float shadow_intensity = 0.3f/(float)shadow_samples;

    unsigned int num_lights = lights.size();
    for(int i = 0; i < num_lights; ++i) {
        BasicLight light = lights[i];
        float Ldist = optix::length(light.pos - hit_point);
        float3 L = optix::normalize(light.pos - hit_point);
        float nDl = optix::dot( p_normal, L);

        intersection.wi_x = optix::dot(L, u_vec);
        intersection.wi_y = optix::dot(L, v_vec);
        intersection.wi_z = optix::dot(L, p_normal);

        for(int i=0;i<shadow_samples;i++){
            float r = light_radius*sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +0,
                (prd_radiance.ray_id.y*shadow_samples +i)*3 +0, 8);
            float theta = sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +1,
                (prd_radiance.ray_id.y*shadow_samples +i)*3 +1, 8);
            float phi   = sampleTEASingle((prd_radiance.ray_id.x*shadow_samples +i)*3 +2, 
                (prd_radiance.ray_id.y*shadow_samples +i)*3 +2, 8);
            float3 offset = make_float3(r*sin(theta)*sin(phi),r*cos(theta)*sin(phi),r*cos(phi));
            float Ldist = optix::length(light.pos + offset - hit_point);
            float3 L = optix::normalize(light.pos + offset - hit_point);
        
            // cast shadow ray
            if ( light.casts_shadow ) {
                shadow_prd.attenuation = make_float3(1.0f);
                optix::Ray shadow_ray = optix::make_Ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
                rtTrace(top_shadower, shadow_ray, shadow_prd);
                color = shadow_prd.attenuation*shadow_intensity*color + (1.f-shadow_intensity)*color;
            }
        }

		
            float3 light_attenuation = shadow_prd.attenuation;

        // If not completely shadowed, light the hit point
        if( fmaxf(light_attenuation) > 0.0f ) {
            wcPatternData pattern_data = wcGetPatternData(intersection,&params);
            float specular = wcEvalSpecular(intersection,pattern_data,&params);

            float3 Lc = light.color * light_attenuation;

            color +=  nDl * Lc * ((1.f - wc_specular_strength) * make_float3(pattern_data.color_r, pattern_data.color_g, pattern_data.color_b) 
                + wc_specular_strength * specular);
        }
    }

    prd_radiance.result = color;
}

